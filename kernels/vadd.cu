
#include <hip/hip_runtime.h>
// filename: vadd.cu
// simple CUDA kernel to add 2 vectors

extern "C"
{
  __global__ void vadd(const float *a, const float *b, float *c)
  {
    int i= threadIdx.x + blockIdx.x * blockDim.x;
    c[i]=a[i]+b[i];
  }
}
